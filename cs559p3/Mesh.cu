#include "hip/hip_runtime.h"
#include "Mesh.h"


#define M_PI 3.14159265358979323846f
#define M_PI_2 1.57079632679489661923f
#define M_PI_4 0.785398163397448309616f

using namespace std;
using namespace glm;

//http://rosettacode.org/wiki/Factors_of_an_integer#C.2B.2B
//we use factors to break problem into sub-problems
int maxFactor(int n, int maxValue)
{
	for(int i = 2; i * i <= n; i++)
		if(n % i == 0)
			if(n / i < maxValue)
				return n / i;
	if(n > maxValue)
		return 1;
	return n;
}
void Mesh::CalculateBlockGridSize(dim3 & block, dim3 & grid, bool iterateFaces, bool heightDoubled)
{
	block = dim3(1, 1, 1);
	if(iterateFaces)
	{
		block.x = maxFactor(width-1, 512);
		block.y = maxFactor(height-1, 512/block.x);
		grid = dim3((width-1)/block.x, (height-1)/block.y, 1);
	}
	else
	{
		block.x = maxFactor(width, 512);
		block.y = maxFactor(height, 512/block.x);
		grid = dim3(width/block.x, height/block.y, 1);
	}
	if(heightDoubled)
		grid.y *= 2;
}


//http://stackoverflow.com/questions/14038589/what-is-the-canonical-way-to-check-for-errors-using-the-cuda-runtime-api
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

//http://stackoverflow.com/questions/4997013/cuda-kernels-consistently-returning-bad-results
void CheckErrorCUDA()
{
	hipError_t cudaResult;
	cudaResult = hipGetLastError();
	if (cudaResult != hipSuccess)
		printf(hipGetErrorString(cudaResult));
}

//This kernel fills the Planar Mesh's Vertex Positions
__global__ void FillPlanarMeshKernel(float3 *pos, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float x_coord = x/((float)(width-1)/2) - 1;
	float y_coord = y/((float)(height-1)/2) - 1;
    pos[y*width+x] = make_float3(x_coord, y_coord, 0.0f);
}


void Mesh::CreatePlanarMesh(int width, int height)
{
	float3* dptr;
	dim3 block, grid;

	hipGraphicsMapResources(1, &resPosition, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, resPosition);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	CalculateBlockGridSize(block, grid);

    FillPlanarMeshKernel<<< grid, block>>>(dptr, width, height);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete

	hipGraphicsUnmapResources(1, &resPosition, 0);
}

//This kernel will create a Sphere Mesh
__global__ void FillSphereMesh(float3 *pos, unsigned int width, unsigned int height, float thetaFac, float phiFac)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float theta, phi;
	float nx, ny, nz;

	theta = y * thetaFac;
	phi = x * phiFac;

	nx = sinf(phi) * cosf(theta);
	ny = sinf(phi) * sinf(theta);
	nz = cosf(phi);

    pos[y*width+x] = make_float3(nx, ny, nz);

	
#if __CUDA_ARCH__ >= 200
	//printf("(%d,%d): Position: (%f,%f,%f)\n", x, y, nx, ny, nz);
#endif
}

void Mesh::CreateSphereMesh()
{
	float3* dptr;
	dim3 block, grid;

	hipGraphicsMapResources(1, &resPosition, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, resPosition);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	CalculateBlockGridSize(block, grid);

	GLfloat thetaFac = (2.0f * M_PI ) / (width-1);
	GLfloat phiFac = M_PI  / (height-1);
    FillSphereMesh<<< grid, block>>>(dptr, width, height, thetaFac, phiFac);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete

	hipGraphicsUnmapResources(1, &resPosition, 0);
}


//This kernel computes the indices for the triangles in the mesh. 
__global__ void FillIndicesKernel(int* indices, int width, int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	//Fill Indices buffer
	int indexOffset = (y*(width-1) + x) * 6;	//every index = 6 vertex indices (2 triangles)
	
	indices[indexOffset++] = (y) * width + (x+1);
	indices[indexOffset++] = (y) * width + x;
	indices[indexOffset++] = (y+1) * width + x;

	indices[indexOffset++] = (y+1) * width + x;
	indices[indexOffset++] = (y+1) * width + (x+1);
	indices[indexOffset++] = y * width + (x+1);
}

void Mesh::CreateIndices() {
	int* ptrIndices;
	dim3 block, grid;

	hipGraphicsMapResources(1, &resIndices, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&ptrIndices, &num_bytes, resIndices);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	CalculateBlockGridSize(block, grid, true);

	FillIndicesKernel<<< grid, block>>>(ptrIndices, width, height);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete

	hipGraphicsUnmapResources(1, &resIndices, 0);
}


//This kernel computes the indices for the triangles in the mesh. 
__global__ void FillWireframeIndicesKernel(int* wireframeIndices, int width, int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	//Fill Wireframe indices buffer
	int wireframeOffset = (y*(width) + x) * 4;	//every index = 4 vertex indices (2 line segments)
	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = y * width + (x+1);
	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = (y+1) * width + x;
}
__global__ void FillWireframeIndicesEdgeCase1Kernel(int* wireframeIndices, int width, int height)
{   //Handle the 'top' edge case
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = height-1;

	//Fill Wireframe indices buffer
	int wireframeOffset = (y*(width) + x) * 4;	//every index = 4 vertex indices (2 line segments)

	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = y * width + (x+1);
}
__global__ void FillWireframeIndicesEdgeCase2Kernel(int* wireframeIndices, int width, int height)
{   //Handle the 'right' edge case
    unsigned int x = width-1;
    unsigned int y = blockIdx.x*blockDim.x + threadIdx.x;	//use x instead of y for the index

	//Fill Wireframe indices buffer
	int wireframeOffset = (y*(width) + x) * 4;	//every index = 4 vertex indices (2 line segments)

	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = (y+1) * width + x;
}

void Mesh::CreateWireframeIndices() {
	int* ptrWireframeIndices;

	hipGraphicsMapResources(1, &resWireframeIndices, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&ptrWireframeIndices, &num_bytes, resWireframeIndices);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(1, 1, 1);
	dim3 grid(width-1, height-1, 1);  //TODO: Fix this so that it uses (width, height) and then handles edge cases inside kernel
	FillWireframeIndicesKernel<<< grid, block>>>(ptrWireframeIndices, width, height);

	//Handle edge cases
	grid.y = 1;
	FillWireframeIndicesEdgeCase1Kernel<<< grid, block>>>(ptrWireframeIndices, width, height);

	grid.x = height-1;
	FillWireframeIndicesEdgeCase2Kernel<<< grid, block>>>(ptrWireframeIndices, width, height);

	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete, only need once since all kernels can be run in parallel

	hipGraphicsUnmapResources(1, &resWireframeIndices, 0);
}

//Compute Level 1.1 atomicAdd workaround (slow)
//https://www.sharcnet.ca/help/index.php/CUDA_tips_and_tricks
__device__ inline void MyAtomicAdd (float *address, float value)
 {
   int oldval, newval, readback;
 
   oldval = __float_as_int(*address);
   newval = __float_as_int(__int_as_float(oldval) + value);
   while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval) 
     {
      oldval = readback;
      newval = __float_as_int(__int_as_float(oldval) + value);
     }
 }

//add b to a atomically
__device__ inline void AtomicAddvec3(glm::vec3* a, glm::vec3* b)
{
#if __CUDA_ARCH__ >= 200
	atomicAdd(&a->x, b->x);
	atomicAdd(&a->y, b->y);
	atomicAdd(&a->z, b->z);
#else
	MyAtomicAdd(&a->x, b->x);
	MyAtomicAdd(&a->y, b->y);
	MyAtomicAdd(&a->z, b->z);
#endif
}


//This kernel computes normals for the mesh by computing the cross product for each triangle, then adding that value to each vertex
__global__ void CalculateNormalsKernel(glm::vec3* positions, glm::vec3* normals, int* indices, unsigned int width)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	unsigned int index = (y * width + x) * 3;

	const vec3 & p1 = positions[indices[index]];
	const vec3 & p2 = positions[indices[index+1]];
	const vec3 & p3 = positions[indices[index+2]];

	vec3 a = p2 - p1;
	vec3 b = p3 - p1;
	vec3 n = glm::cross(a,b);
	if(length(n) > 0.0f)
		n =  glm::normalize(n);
	n = -n;		//why do we need this??

	AtomicAddvec3(&normals[indices[index]], &n);
	AtomicAddvec3(&normals[indices[index+1]], &n);
	AtomicAddvec3(&normals[indices[index+2]], &n);

}

__global__ void NormalizeNormals(glm::vec3* normals, unsigned int width)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	unsigned int index = y * width + x;

	if(glm::length(normals[index]) > 0.0f)
		normals[index] = glm::normalize(normals[index]);
	else
		normals[index] = vec3(0,0,-10);

#if __CUDA_ARCH__ >= 200
	//printf("(%d,%d): Normal: (%f,%f,%f)\n", x, y, normals[index].x, normals[index].y, normals[index].z);
#endif
}

__global__ void ZeroBufferVec3(glm::vec3* buffer, unsigned int width)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	buffer[y * width + x] = vec3(0.0f);
}


void Mesh::CalculateNormals()
{
	size_t num_bytes;
	int* indices;
	vec3* positions, *normals;
	dim3 block, grid;

	//Map resources and get pointers
	hipGraphicsMapResources(1, &resIndices, 0);
	hipGraphicsResourceGetMappedPointer((void **)&indices, &num_bytes, resIndices);
	hipGraphicsMapResources(1, &resPosition, 0);
	hipGraphicsResourceGetMappedPointer((void **)&positions, &num_bytes, resPosition);
	hipGraphicsMapResources(1, &resNormals, 0);
	hipGraphicsResourceGetMappedPointer((void **)&normals, &num_bytes, resNormals);

	//First, zero normals
	CalculateBlockGridSize(block, grid);
	ZeroBufferVec3<<< grid, block>>>(normals, width);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete
	
	//Second, generate averaged normals based on faces
	CalculateBlockGridSize(block, grid, true, true);
	CalculateNormalsKernel<<< grid, block>>>(positions, normals, indices, width-1);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete
	CheckErrorCUDA();
	
    //Lastly, Normalize
	CalculateBlockGridSize(block, grid);
	NormalizeNormals<<< grid, block>>>(normals, width);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete

	//Unmap Resources
	hipGraphicsUnmapResources(1, &resNormals, 0);
	hipGraphicsUnmapResources(1, &resPosition, 0);
	hipGraphicsUnmapResources(1, &resIndices, 0);
}

//This kernel creates a VBO of Positions that will draw Normal geometry lines
__global__ void CreateNormalsVisualizationKernel(glm::vec3* positions, glm::vec3* normals, glm::vec3* normalPositions, unsigned int width)
{
	const float normalScalar = 0.05f;

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	unsigned int index = (y * width + x) * 2;

	glm::vec3& position = positions[y * width + x];
	glm::vec3& normal = normals[y * width + x];

	normalPositions[index] = position;
	normalPositions[index+1] = position + normal*normalScalar;
	
#if __CUDA_ARCH__ >= 200
	/*printf("(%d,%d): Positions: (%f,%f,%f) (%f,%f,%f)\n", x, y,
		normalPositions[index].x, normalPositions[index].y, normalPositions[index].z,
		normalPositions[index+1].x, normalPositions[index+1].y, normalPositions[index+1].z);*/
#endif
}


void Mesh::CreateNormalsVisualization()
{
	size_t num_bytes;
	vec3* positions, *normals, *normalPositions;
	dim3 block, grid;

	hipGraphicsMapResources(1, &resPosition, 0);
	hipGraphicsMapResources(1, &resNormals, 0);
	hipGraphicsMapResources(1, &resNormalPositions, 0);
	hipGraphicsResourceGetMappedPointer((void **)&positions, &num_bytes, resPosition);
	hipGraphicsResourceGetMappedPointer((void **)&normals, &num_bytes, resNormals);
	hipGraphicsResourceGetMappedPointer((void **)&normalPositions, &num_bytes, resNormalPositions);
	
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	CalculateBlockGridSize(block, grid);
	CreateNormalsVisualizationKernel<<< grid, block>>>(positions, normals, normalPositions, width);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete
	
	hipGraphicsUnmapResources(1, &resNormalPositions, 0);
	hipGraphicsUnmapResources(1, &resNormals, 0);
	hipGraphicsUnmapResources(1, &resPosition, 0);
}

//Initialize default values
Mesh::Mesh(void)
{
	width = height = 0;

	vao = GL_BAD_VALUE;

	vboPosition = GL_BAD_VALUE;
	resPosition = NULL;
	vboNormals = GL_BAD_VALUE;
	resNormals = NULL;

	vboIndices = GL_BAD_VALUE;
	numIndices = 0;
	resIndices = NULL;

	vboWireframeIndices = GL_BAD_VALUE;
	numWireframeIndices = 0;
	resWireframeIndices = NULL;

	vboNormalPositions = GL_BAD_VALUE;
	numNormalPositions = 0;
	resNormalPositions = NULL;

	Ka = vec3(1.0f);
	Ks = vec3(1.0f);
	Kd = vec3(1.0f);
	Shininess = 100.0f;

	wireframeMode = false;
	drawNormals = false;
}


Mesh::~Mesh(void)
{
	//TODO: Add handle asserts here to make sure objects are de-allocated.  example:
	//assert(vao == GL_BAD_VALUE);
}



void Mesh::Initialize(int width, int height)
{
	//store the Mesh width and height
	this->width = width;
	this->height = height;

	//calculate the number of indices the Mesh will have
	this->numIndices = (width-1) * (height-1) * 6;
	this->numWireframeIndices = (width) * (height) * 4;
	this->numNormalPositions = width * height * 2;
	
	//Create VAO
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);

//Create Position VBO
	glGenBuffers(1, &vboPosition);
	glBindBuffer(GL_ARRAY_BUFFER, vboPosition);
	size_t bytesPosition = sizeof(vec3) * width * height;
	glBufferData(GL_ARRAY_BUFFER, bytesPosition, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resPosition, vboPosition, cudaGraphicsMapFlagsWriteDiscard);
//Associate with Shader layout element 0, enable
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(vec3(1.0f)), 0);
	glEnableVertexAttribArray(0);

//Create Normal VBO
	glGenBuffers(1, &vboNormals);
	glBindBuffer(GL_ARRAY_BUFFER, vboNormals);
	glBufferData(GL_ARRAY_BUFFER, bytesPosition, 0, GL_STATIC_DRAW);//same size as Position VBO, both are vec3
	
    hipGraphicsGLRegisterBuffer(&resNormals, vboNormals, cudaGraphicsMapFlagsWriteDiscard);
	//Associate with Shader layout element 1, enable
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(vec3(1.0f)), 0);
	glEnableVertexAttribArray(1);

	//Unbind array buffer and vao
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);


//Create Indices VBO
	glGenBuffers(1, &vboIndices);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboIndices);
	size_t bytesIndices = numIndices * sizeof(GLuint);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, bytesIndices, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resIndices, vboIndices, cudaGraphicsMapFlagsWriteDiscard);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);    //Unbind array element buffer
	

//Create Wireframe Indices VBO
	glGenBuffers(1, &vboWireframeIndices);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboWireframeIndices);
	bytesIndices = numWireframeIndices * sizeof(GLuint);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, bytesIndices, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resWireframeIndices, vboWireframeIndices, cudaGraphicsMapFlagsWriteDiscard);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);    //Unbind array element buffer
	

//Create Normal Visualization VAO
	glGenVertexArrays(1, &vaoNormalPositions);
	glBindVertexArray(vaoNormalPositions);

//Create Normal Visualization VBO - do we need a VAO? yes
	glGenBuffers(1, &vboNormalPositions);
	glBindBuffer(GL_ARRAY_BUFFER, vboNormalPositions);
	size_t bytesNormalPositions = numNormalPositions * sizeof(vec3(1.0f));
	glBufferData(GL_ARRAY_BUFFER, bytesNormalPositions, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resNormalPositions, vboNormalPositions, cudaGraphicsMapFlagsWriteDiscard);
//Associate with Shader layout element 0, enable
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(vec3(1.0f)), 0);
	glEnableVertexAttribArray(0);

//Unbind array buffer and vao
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

	
	//Create basic planar mesh and indices
	CreatePlanarMesh(width, height);
	//CreateSphereMesh();

	CreateIndices();
	CreateWireframeIndices();

	//some test fucntions:
	CalculateNormals();
	CreateNormalsVisualization();
}

void Mesh::Draw()
{
	glBindVertexArray(vao);
	if(wireframeMode) {
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboWireframeIndices);
		glDrawElements(GL_LINES, numWireframeIndices, GL_UNSIGNED_INT, (GLvoid*)0);
	} else {
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboIndices);
		glDrawElements(GL_TRIANGLES, numIndices*8/8, GL_UNSIGNED_INT, (GLvoid*)0);
	}
	glBindVertexArray(0);

}

void Mesh::DrawNormals()
{
	glBindVertexArray(vaoNormalPositions);
	glDrawArrays(GL_LINES, 0, numNormalPositions);
	glBindVertexArray(0);
}



void Mesh::TakeDown()
{
	//TODO: De-Allocate GL objects here
}
