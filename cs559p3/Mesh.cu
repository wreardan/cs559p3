#include "hip/hip_runtime.h"
#include "Mesh.h"


#define M_PI 3.14159265358979323846
#define M_PI_2 1.57079632679489661923
#define M_PI_4 0.785398163397448309616

using namespace glm;

//This kernel fills the Planar Mesh's Vertex Positions
__global__ void FillPlanarMeshKernel(float3 *pos, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float x_coord = x/((float)(width-1)/2) - 1;
	float y_coord = y/((float)(height-1)/2) - 1;
    pos[y*width+x] = make_float3(x_coord, y_coord, 0.0f);
}


void Mesh::CreatePlanarMesh(int width, int height)
{
	float3* dptr;

	hipGraphicsMapResources(1, &resPosition, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, resPosition);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(8, 8, 1);
	if(width % 8 || height % 8)
	{
		block.x = 1; block.y = 1;
	}
    dim3 grid(width / block.x, height / block.y, 1);

    FillPlanarMeshKernel<<< grid, block>>>(dptr, width, height);

	hipGraphicsUnmapResources(1, &resPosition, 0);
}

//This kernel will create a Sphere Mesh
__global__ void FillSphereMesh(float3 *pos, unsigned int width, unsigned int height, float thetaFac, float phiFac)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float theta, phi;
	float nx, ny, nz, s, t;

	theta = y * thetaFac;
    s = y / (float)width;
	phi = x * phiFac;
    t = x / (float)height;

	nx = sinf(phi) * cosf(theta);
	ny = sinf(phi) * sinf(theta);
	nz = cosf(phi);

    pos[y*width+x] = make_float3(nx, ny, nz);

	
#if __CUDA_ARCH__ >= 200
	//printf("(%d,%d): Position: (%f,%f,%f)\n", x, y, nx, ny, nz);
#endif
}

void Mesh::CreateSphereMesh()
{
	float3* dptr;

	hipGraphicsMapResources(1, &resPosition, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, resPosition);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(4, 4, 1);
	if(width % 4 || height % 4)
	{
		block.x = 1; block.y = 1;
	}
    dim3 grid(width / block.x, height / block.y, 1);
	

	GLfloat thetaFac = (2.0f * M_PI ) / (width-1);
	GLfloat phiFac = M_PI  / (height-1);
    FillSphereMesh<<< grid, block>>>(dptr, width, height, thetaFac, phiFac);

	hipGraphicsUnmapResources(1, &resPosition, 0);
}


//This kernel computes the indices for the triangles in the mesh. 
__global__ void FillIndicesKernel(int* indices, int width, int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	//Fill Indices buffer
	int indexOffset = (y*(width-1) + x) * 6;	//every index = 6 vertex indices (2 triangles)
	
	indices[indexOffset++] = (y) * width + (x+1);
	indices[indexOffset++] = (y) * width + x;
	indices[indexOffset++] = (y+1) * width + x;

	indices[indexOffset++] = (y+1) * width + x;
	indices[indexOffset++] = (y+1) * width + (x+1);
	indices[indexOffset++] = y * width + (x+1);
}

void Mesh::CreateIndices() {
	int* ptrIndices;

	hipGraphicsMapResources(1, &resIndices, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&ptrIndices, &num_bytes, resIndices);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(1, 1, 1);
    dim3 grid(width-1, height-1, 1);

	FillIndicesKernel<<< grid, block>>>(ptrIndices, width, height);

	hipGraphicsUnmapResources(1, &resIndices, 0);
}


//This kernel computes the indices for the triangles in the mesh. 
__global__ void FillWireframeIndicesKernel(int* wireframeIndices, int width, int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	//Fill Wireframe indices buffer
	int wireframeOffset = (y*(width-1) + x) * 4;	//every index = 4 vertex indices (2 line segments)
	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = y * width + (x+1);
	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = (y+1) * width + x;
}
__global__ void FillWireframeIndicesEdgeCasesKernel(int* wireframeIndices, int width, int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	//Fill Wireframe indices buffer
	int wireframeOffset = (y*(width-1) + x) * 4;	//every index = 4 vertex indices (2 line segments)
	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = y * width + (x+1);
	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = (y+1) * width + x;
}

void Mesh::CreateWireframeIndices() {
	int* ptrWireframeIndices;

	hipGraphicsMapResources(1, &resWireframeIndices, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&ptrWireframeIndices, &num_bytes, resWireframeIndices);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(1, 1, 1);
	dim3 grid(width-1, height-1, 1);  //TODO: Fix this so that it uses (width, height) and then handles edge cases inside kernel

	FillWireframeIndicesKernel<<< grid, block>>>(ptrWireframeIndices, width, height);

	hipGraphicsUnmapResources(1, &resWireframeIndices, 0);
}

//This kernel computes normals for the mesh
__global__ void CalculateNormalsKernel(glm::vec3* positions, glm::vec3* normals, int* indices)
{
    unsigned int index = (blockIdx.x*blockDim.x + threadIdx.x) * 3;

	

	const vec3 & p1 = positions[indices[index]];
	const vec3 & p2 = positions[indices[index+1]];
	const vec3 & p3 = positions[indices[index+2]];

	vec3 a = p2 - p1;
	vec3 b = p3 - p1;
	vec3 n = glm::cross(a,b);
	if(length(n) > 0.0f)
		n =  glm::normalize(n);
	n = -n;

#if __CUDA_ARCH__ >= 200

	atomicAdd(&normals[indices[index]].x, n.x);
	atomicAdd(&normals[indices[index]].y, n.y);
	atomicAdd(&normals[indices[index]].z, n.z);

	atomicAdd(&normals[indices[index+1]].x, n.x);
	atomicAdd(&normals[indices[index+1]].y, n.y);
	atomicAdd(&normals[indices[index+1]].z, n.z);

	atomicAdd(&normals[indices[index+2]].x, n.x);
	atomicAdd(&normals[indices[index+2]].y, n.y);
	atomicAdd(&normals[indices[index+2]].z, n.z);
//#else

	
	//normals[indices[index]] += n;  //Does this need to be atomic?  i.e. atomicAdd()? see above code for possible solution
	//normals[indices[index+1]] += n;
	//normals[indices[index+2]] += n;

	/*printf("%d: (%f,%f,%f), (%f,%f,%f), (%f,%f,%f)\n", index/3,
		normals[indices[index]].x, normals[indices[index]].y, normals[indices[index]].z,
		normals[indices[index+1]].x, normals[indices[index+1]].y, normals[indices[index+1]].z,
		normals[indices[index+2]].x, normals[indices[index+2]].y, normals[indices[index+2]].z);*/

	//printf("Block (%d): Normal: (%f,%f,%f) ", index/3, n.x, n.y, n.z);
#endif

}

__global__ void NormalizeNormals(glm::vec3* normals, unsigned int width)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	unsigned int index = y * width + x;

	if(glm::length(normals[index]) > 0.0f)
		normals[index] = glm::normalize(normals[index]);

#if __CUDA_ARCH__ >= 200
	//printf("(%d,%d): Normal: (%f,%f,%f)\n", x, y, normals[index].x, normals[index].y, normals[index].z);
#endif
}

__global__ void ZeroBufferVec3(glm::vec3* buffer, unsigned int width)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	buffer[y * width + x] = vec3(0.0f);
}


void Mesh::CalculateNormals()
{
	size_t num_bytes;
	int* indices;
	vec3* positions, *normals;

	//Map resources and get pointers
	hipGraphicsMapResources(1, &resIndices, 0);
	hipGraphicsResourceGetMappedPointer((void **)&indices, &num_bytes, resIndices);
	hipGraphicsMapResources(1, &resPosition, 0);
	hipGraphicsResourceGetMappedPointer((void **)&positions, &num_bytes, resPosition);
	hipGraphicsMapResources(1, &resNormals, 0);
	hipGraphicsResourceGetMappedPointer((void **)&normals, &num_bytes, resNormals);

	//First, zero normals
	dim3 block(8, 8, 1);
	if(width % 8 || height % 8)
	{
		block.x = 1; block.y = 1;
	}
    dim3 grid(width / block.x, height / block.y, 1);
	ZeroBufferVec3<<< grid, block>>>(normals, width);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete
	
	//Second, generate averaged normals based on faces
	block = dim3(1,1,1);
    grid = dim3(numIndices / 3 / block.x, 1, 1);
	CalculateNormalsKernel<<< grid, block>>>(positions, normals, indices);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete

	
    //Lastly, Normalize
	block = dim3(8,8,1);
	if(width % 8 || height % 8)
	{
		block.x = 1; block.y = 1;
	}
    grid = dim3(width / block.x, height / block.y, 1);
	NormalizeNormals<<< grid, block>>>(normals, width);
	hipDeviceSynchronize();  //Wait for CUDA kernel to Complete

	//Unmap Resources
	hipGraphicsUnmapResources(1, &resNormals, 0);
	hipGraphicsUnmapResources(1, &resPosition, 0);
	hipGraphicsUnmapResources(1, &resIndices, 0);
}

//This kernel creates a VBO of Positions that will draw Normal geometry lines
__global__ void CreateNormalsVisualizationKernel(glm::vec3* positions, glm::vec3* normals, glm::vec3* normalPositions,
												 unsigned int * normalIndices, unsigned int width)
{
	const float normalScalar = 0.05f;

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	unsigned int index = (y * width + x) * 2;

	glm::vec3& position = positions[y * width + x];
	glm::vec3& normal = normals[y * width + x];

	normalPositions[index] = position;
	normalPositions[index+1] = position + normal*normalScalar;

	normalIndices[index] = index;
	normalIndices[index+1] = index+1;
	
#if __CUDA_ARCH__ >= 200
	/*printf("(%d,%d): Positions: (%f,%f,%f) (%f,%f,%f)\n", x, y,
		normalPositions[index].x, normalPositions[index].y, normalPositions[index].z,
		normalPositions[index+1].x, normalPositions[index+1].y, normalPositions[index+1].z);*/
#endif
}


void Mesh::CreateNormalsVisualization()
{
	size_t num_bytes;
	vec3* positions, *normals, *normalPositions;
	unsigned int * normalIndices;

	hipGraphicsMapResources(1, &resPosition, 0);
	hipGraphicsMapResources(1, &resNormals, 0);
	hipGraphicsMapResources(1, &resNormalPositions, 0);
	hipGraphicsMapResources(1, &resNormalIndices, 0);
	hipGraphicsResourceGetMappedPointer((void **)&positions, &num_bytes, resPosition);
	hipGraphicsResourceGetMappedPointer((void **)&normals, &num_bytes, resNormals);
	hipGraphicsResourceGetMappedPointer((void **)&normalPositions, &num_bytes, resNormalPositions);
	hipGraphicsResourceGetMappedPointer((void **)&normalIndices, &num_bytes, resNormalIndices);
	
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(8, 8, 1);
	if(width % 8 || height % 8)
	{
		block.x = 1; block.y = 1;
	}
    dim3 grid(width / block.x, height / block.y, 1);

	CreateNormalsVisualizationKernel<<< grid, block>>>(positions, normals, normalPositions, normalIndices, width);
	
	hipGraphicsUnmapResources(1, &resNormalIndices, 0);
	hipGraphicsUnmapResources(1, &resNormalPositions, 0);
	hipGraphicsUnmapResources(1, &resNormals, 0);
	hipGraphicsUnmapResources(1, &resPosition, 0);
}



using namespace std;
using namespace glm;

//Initialize default values
Mesh::Mesh(void)
{
	width = height = 0;

	vao = GL_BAD_VALUE;

	vboPosition = GL_BAD_VALUE;
	resPosition = NULL;
	vboNormals = GL_BAD_VALUE;
	resNormals = NULL;

	vboIndices = GL_BAD_VALUE;
	numIndices = 0;
	resIndices = NULL;

	vboWireframeIndices = GL_BAD_VALUE;
	numWireframeIndices = 0;
	resWireframeIndices = NULL;

	vboNormalPositions = GL_BAD_VALUE;
	numNormalPositions = 0;
	resNormalPositions = NULL;

	Ka = vec3(1.0f);
	Ks = vec3(1.0f);
	Kd = vec3(1.0f);
	Shininess = 100.0f;

	wireframeMode = false;
	drawNormals = false;
}


Mesh::~Mesh(void)
{
	//TODO: Add handle asserts here to make sure objects are de-allocated.  example:
	//assert(vao == GL_BAD_VALUE);
}



void Mesh::Initialize(int width, int height)
{
	//store the Mesh width and height
	this->width = width;
	this->height = height;

	//calculate the number of indices the Mesh will have
	this->numIndices = (width-1) * (height-1) * 6;
	this->numWireframeIndices = (width-1) * (height-1) * 4;
	this->numNormalPositions = width * height * 2;
	
	//Create VAO
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);

//Create Position VBO
	glGenBuffers(1, &vboPosition);
	glBindBuffer(GL_ARRAY_BUFFER, vboPosition);
	size_t bytesPosition = sizeof(vec3) * width * height;
	glBufferData(GL_ARRAY_BUFFER, bytesPosition, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resPosition, vboPosition, cudaGraphicsMapFlagsWriteDiscard);
//Associate with Shader layout element 0, enable
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(vec3(1.0f)), 0);
	glEnableVertexAttribArray(0);

//Create Normal VBO
	glGenBuffers(1, &vboNormals);
	glBindBuffer(GL_ARRAY_BUFFER, vboNormals);
	glBufferData(GL_ARRAY_BUFFER, bytesPosition, 0, GL_STATIC_DRAW);//same size as Position VBO, both are vec3
	
    hipGraphicsGLRegisterBuffer(&resNormals, vboNormals, cudaGraphicsMapFlagsWriteDiscard);
	//Associate with Shader layout element 1, enable
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(vec3(1.0f)), 0);
	glEnableVertexAttribArray(1);

	//Unbind array buffer and vao
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);


//Create Indices VBO
	glGenBuffers(1, &vboIndices);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboIndices);
	size_t bytesIndices = numIndices * sizeof(GLuint);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, bytesIndices, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resIndices, vboIndices, cudaGraphicsMapFlagsWriteDiscard);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);    //Unbind array element buffer
	

//Create Wireframe Indices VBO
	glGenBuffers(1, &vboWireframeIndices);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboWireframeIndices);
	bytesIndices = numWireframeIndices * sizeof(GLuint);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, bytesIndices, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resWireframeIndices, vboWireframeIndices, cudaGraphicsMapFlagsWriteDiscard);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);    //Unbind array element buffer
	

//Create Normal Visualization VAO
	glGenVertexArrays(1, &vaoNormalPositions);
	glBindVertexArray(vaoNormalPositions);

//Create Normal Visualization VBO - do we need a VAO? yes
	glGenBuffers(1, &vboNormalPositions);
	glBindBuffer(GL_ARRAY_BUFFER, vboNormalPositions);
	size_t bytesNormalPositions = sizeof(vec3(1.0f)) * width * height * 2;
	glBufferData(GL_ARRAY_BUFFER, bytesNormalPositions, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resNormalPositions, vboNormalPositions, cudaGraphicsMapFlagsWriteDiscard);
//Associate with Shader layout element 0, enable
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(vec3(1.0f)), 0);
	glEnableVertexAttribArray(0);

//Unbind array buffer and vao
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);

//Create Normal Indices VBO
	glGenBuffers(1, &vboNormalIndices);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboNormalIndices);
	size_t bytesNormalIndices = numNormalPositions * sizeof(GLuint);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, bytesNormalIndices, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resNormalIndices, vboNormalIndices, cudaGraphicsMapFlagsWriteDiscard);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, 0);    //Unbind array element buffer

	
	//Create basic planar mesh and indices
	//CreatePlanarMesh(width, height);
	CreateSphereMesh();

	CreateIndices();
	CreateWireframeIndices();

	//some test fucntions:
	CalculateNormals();
	hipDeviceSynchronize();	//http://stackoverflow.com/questions/15669841/cuda-hello-world-printf-not-working-even-with-arch-sm-20
	CreateNormalsVisualization();
}

void Mesh::Draw()
{
	glBindVertexArray(vao);
	if(wireframeMode) {
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboWireframeIndices);
		glDrawElements(GL_LINES, numWireframeIndices, GL_UNSIGNED_INT, (GLvoid*)0);
	} else {
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboIndices);
		glDrawElements(GL_TRIANGLES, numIndices*8/8, GL_UNSIGNED_INT, (GLvoid*)0);
	}
	glBindVertexArray(0);

}

void Mesh::DrawNormals()
{
	glBindVertexArray(vaoNormalPositions);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboNormalIndices);
	glDrawElements(GL_LINES, numNormalPositions, GL_UNSIGNED_INT, (GLvoid*)0);
	glBindVertexArray(0);
}



void Mesh::TakeDown()
{
	//TODO: De-Allocate GL objects here
}
