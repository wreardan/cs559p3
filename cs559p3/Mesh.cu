#include "hip/hip_runtime.h"
#include "Mesh.h"


#define M_PI 3.14159265358979323846
#define M_PI_2 1.57079632679489661923
#define M_PI_4 0.785398163397448309616

using namespace glm;

//This kernel fills the Planar Mesh's Vertex Positions
__global__ void FillPlanarMeshKernel(float3 *pos, unsigned int width, unsigned int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	float x_coord = x/((float)(width-1)/2) - 1;
	float y_coord = y/((float)(height-1)/2) - 1;
    pos[y*width+x] = make_float3(x_coord, y_coord, 0.0f);
}


void Mesh::CreatePlanarMesh(int width, int height)
{
	float3* dptr;

	hipGraphicsMapResources(1, &resPosition, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, resPosition);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(8, 8, 1);
	if(width % 8 || height % 8)
		block.x = 1; block.y = 1;
    dim3 grid(width / block.x, height / block.y, 1);

    FillPlanarMeshKernel<<< grid, block>>>(dptr, width, height);

	hipGraphicsUnmapResources(1, &resPosition, 0);
}

//This kernel will create a Sphere Mesh
__global__ void FillSphereMesh(float3 *pos, unsigned int width, unsigned int height, float thetaFac, float phiFac)
{

    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	GLfloat theta, phi;
	GLfloat nx, ny, nz, s, t;

	theta = y * thetaFac;
    s = (GLfloat)y / width;
	phi = x * phiFac;
    t = (GLfloat)x / height;

	nx = sinf(phi) * cosf(theta);
	ny = sinf(phi) * sinf(theta);
	nz = cosf(phi);

    pos[y*width+x] = make_float3(nx, ny, nz);
}

void Mesh::CreateSphereMesh(int width, int height)
{
	float3* dptr;

	hipGraphicsMapResources(1, &resPosition, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&dptr, &num_bytes, resPosition);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(8, 8, 1);
	if(width % 8 || height % 8)
		block.x = 1; block.y = 1;
    dim3 grid(width / block.x, height / block.y, 1);
	

	GLfloat thetaFac = (2.0f * M_PI ) / (width-1);
	GLfloat phiFac = M_PI  / (height-1);
    FillSphereMesh<<< grid, block>>>(dptr, width, height, thetaFac, phiFac);

	hipGraphicsUnmapResources(1, &resPosition, 0);
}


//This kernel computes the indices for the triangles in the mesh. 
__global__ void FillIndicesKernel(int* indices, int* wireframeIndices, int width, int height)
{
    unsigned int x = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y*blockDim.y + threadIdx.y;

	//Fill Indices buffer
	int indexOffset = (y*(width-1) + x) * 6;	//every index = 6 vertex indices (2 triangles)
	
	indices[indexOffset++] = (y) * width + (x+1);
	indices[indexOffset++] = (y) * width + x;
	indices[indexOffset++] = (y+1) * width + x;

	indices[indexOffset++] = (y+1) * width + x;
	indices[indexOffset++] = (y+1) * width + (x+1);
	indices[indexOffset++] = y * width + (x+1);

	//Fill Wireframe indices buffer
	int wireframeOffset = (y*(width-1) + x) * 4;	//every index = 4 vertex indices (2 line segments)
	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = y * width + (x+1);
	wireframeIndices[wireframeOffset++] = y * width + x;
	wireframeIndices[wireframeOffset++] = (y+1) * width + x;
}

void Mesh::CreateIndices(int width, int height) {
	int* ptrIndices, *ptrWireframeIndices;

	hipGraphicsMapResources(1, &resIndices, 0);
	hipGraphicsMapResources(1, &resWireframeIndices, 0);
	size_t num_bytes;
	hipGraphicsResourceGetMappedPointer((void **)&ptrIndices, &num_bytes, resIndices);
	hipGraphicsResourceGetMappedPointer((void **)&ptrWireframeIndices, &num_bytes, resWireframeIndices);
	
    // execute the kernel - performance improvement when width and height are divisible by 8
	dim3 block(1, 1, 1);
    dim3 grid(width-1, height-1, 1);

	FillIndicesKernel<<< grid, block>>>(ptrIndices, ptrWireframeIndices, width, height);

	hipGraphicsUnmapResources(1, &resIndices, 0);
	hipGraphicsUnmapResources(1, &resWireframeIndices, 0);
}

//This kernel computes normals for the mesh
__global__ void CalculateNormalsKernel(glm::vec3* positions, glm::vec3* normals, int* indices, int width, int height)
{
	
}

void Mesh::CalculateNormals()
{
	size_t num_bytes;
	int* indices;
	vec3* positions, *normals;

	hipGraphicsMapResources(1, &resIndices, 0);
	hipGraphicsMapResources(1, &resPosition, 0);
	//hipGraphicsMapResources(1, &resNormals, 0);
	hipGraphicsResourceGetMappedPointer((void **)&indices, &num_bytes, resIndices);
	
	dim3 block(8, 8, 1);
	if(width % 8 || height % 8)
		block.x = 1; block.y = 1;
    dim3 grid(width / block.x, height / block.y, 1);

	CalculateNormalsKernel<<< grid, block>>>(positions, normals, indices, width, height);

	hipGraphicsUnmapResources(1, &resIndices, 0);
}


using namespace std;
using namespace glm;

//Initialize default values
Mesh::Mesh(void)
{
	width = height = 0;

	vao = GL_BAD_VALUE;

	vboPosition = GL_BAD_VALUE;
	resPosition = NULL;
	vboNormals = GL_BAD_VALUE;
	resNormals = NULL;

	vboIndices = GL_BAD_VALUE;
	numIndices = 0;
	resIndices = NULL;

	vboWireframeIndices = GL_BAD_VALUE;
	numWireframeIndices = 0;
	resWireframeIndices = NULL;

	vboNormalPositions = GL_BAD_VALUE;
	numNormalPositions = 0;
	resNormalPositions = NULL;

	Ka = vec3(1.0f);
	Ks = vec3(1.0f);
	Kd = vec3(1.0f);
	Shininess = 100.0f;

	wireframeMode = false;
}


Mesh::~Mesh(void)
{
	//TODO: Add handle asserts here to make sure objects are de-allocated.  example:
	//assert(vao == GL_BAD_VALUE);
}



void Mesh::Initialize(int width, int height)
{
	//store the Mesh width and height
	this->width = width;
	this->height = height;

	//calculate the number of indices the Mesh will have
	this->numIndices = (width-1) * (height-1) * 6;
	this->numWireframeIndices = (width-1) * (height-1) * 4;
	
	//Create VAO
	glGenVertexArrays(1, &vao);
	glBindVertexArray(vao);

//Create Position VBO
	glGenBuffers(1, &vboPosition);
	glBindBuffer(GL_ARRAY_BUFFER, vboPosition);
	size_t bytesPosition = sizeof(vec3) * width * height;
	glBufferData(GL_ARRAY_BUFFER, bytesPosition, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resPosition, vboPosition, cudaGraphicsMapFlagsWriteDiscard);
//Associate with Shader layout element 0, enable
	glVertexAttribPointer(0, 3, GL_FLOAT, GL_FALSE, sizeof(vec3(1.0f)), 0);
	glEnableVertexAttribArray(0);

//Create Normal VBO
	glGenBuffers(1, &vboNormals);
	glBindBuffer(GL_ARRAY_BUFFER, vboNormals);
	glBufferData(GL_ARRAY_BUFFER, bytesPosition, 0, GL_STATIC_DRAW);//same size as Position VBO, both are vec3
	
    hipGraphicsGLRegisterBuffer(&resPosition, vboPosition, cudaGraphicsMapFlagsWriteDiscard);
//Associate with Shader layout element 1, enable
	glVertexAttribPointer(1, 3, GL_FLOAT, GL_FALSE, sizeof(vec3(1.0f)), 0);
	glEnableVertexAttribArray(1);

//Create Indices VBO
	glGenBuffers(1, &vboIndices);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboIndices);
	size_t bytesIndices = numIndices * sizeof(GLuint);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, bytesIndices, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resIndices, vboIndices, cudaGraphicsMapFlagsWriteDiscard);

//Unbind array buffer and vao
	glBindBuffer(GL_ARRAY_BUFFER, 0);
	glBindVertexArray(0);
	
//Create Wireframe VBO
	glGenBuffers(1, &vboWireframeIndices);
	glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboWireframeIndices);
	bytesIndices = numWireframeIndices * sizeof(GLuint);
	glBufferData(GL_ELEMENT_ARRAY_BUFFER, bytesIndices, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resWireframeIndices, vboWireframeIndices, cudaGraphicsMapFlagsWriteDiscard);

//Create Normal Visualization VBO
	glGenBuffers(1, &vboNormalPositions);
	glBindBuffer(GL_ARRAY_BUFFER, vboNormalPositions);
	size_t bytesNormalPositions = sizeof(vec3(1.0f)) * width * height * 2;
	glBufferData(GL_ARRAY_BUFFER, bytesNormalPositions, 0, GL_STATIC_DRAW);
	
    hipGraphicsGLRegisterBuffer(&resPosition, vboPosition, cudaGraphicsMapFlagsWriteDiscard);

	
	//Create basic planar mesh and indices
	//CreatePlanarMesh(width, height);
	CreateSphereMesh(width, height);
	CreateIndices(width, height);
}

void Mesh::Draw()
{
	glBindVertexArray(vao);
	if(wireframeMode) {
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboWireframeIndices);
		glDrawElements(GL_LINES, numWireframeIndices, GL_UNSIGNED_INT, (GLvoid*)0);
	} else {
		glBindBuffer(GL_ELEMENT_ARRAY_BUFFER, vboIndices);
		glDrawElements(GL_TRIANGLES, numIndices*8/8, GL_UNSIGNED_INT, (GLvoid*)0);
	}
	glBindVertexArray(0);
}

void Mesh::TakeDown()
{
	//TODO: De-Allocate GL objects here
}
